#include "hip/hip_runtime.h"
/**
  All-CUDA source code for 1D Barnes-Hut

*/
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>

#include "barnes1d_cudatree.h"

__global__ void tryTraverse(const BarnesNodeData *treeNodes,int firstLeaf,int nTreeNodes,float *acc) {
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	if (i>=firstLeaf && i<nTreeNodes) {
#ifdef CUDA_USE_RECURSION
		BarnesParaTree tree(treeNodes,firstLeaf,nTreeNodes);
#else
		BarnesParaTree untertree(treeNodes,firstLeaf,nTreeNodes);
		ManualStackTree<BarnesKey,typeof(untertree)> tree(untertree);
#endif
		BarnesKey treeRoot=1;
		BarnesConsumer<typeof(tree),BarnesKey> c(tree,treeNodes[i]);
		
		// Expand the tree root into the consumer
		tree.requestNode(treeRoot,c);

#ifndef CUDA_USE_RECURSION	
		tree.iterateToConsumer(c);
#endif
		
		acc[i]=c.acc;
	}
}


#define check(cudacall) { int err=cudacall; if (err!=hipSuccess) std::cout<<"CUDA ERROR "<<err<<" at line "<<__LINE__<<"'s "<<#cudacall<<"\n";}

int main() {
	// Build tree on CPU:
	std::vector<BarnesNodeData> n;
	n.push_back(BarnesNodeData(-666.0,-666.0,0.0,100.0)); // invalid entry 0
	
	n.push_back(BarnesNodeData(10.0,50.0,0.0,100.0)); // [1] root
	
	n.push_back(BarnesNodeData(3.0,25.0,0.0,50.0));	// first level
	n.push_back(BarnesNodeData(7.0,75.0,50.0,100.0));
	
	BarnesKey firstLeaf=n.size();
	
	n.push_back(BarnesNodeData(1.0,10.0,0.0,25));  // leaf level
	n.push_back(BarnesNodeData(2.0,40.0,25.0,50.0));
	n.push_back(BarnesNodeData(4.0,60.0,50.0,75));
	n.push_back(BarnesNodeData(3.0,85.0,75.0,100.0));
	
	
	// Copy tree to GPU
	BarnesNodeData *gn=0;
	check(hipMalloc((void **)&gn, n.size()*sizeof(n[0])));
	check(hipMemcpy(gn,&n[0],n.size()*sizeof(n[0]),hipMemcpyHostToDevice));
	
	// Accelerations for each leaf:
	float *gacc=0;
	check(hipMalloc((void **)&gacc, n.size()*sizeof(float)));
	
	// Walk tree on GPU
	tryTraverse<<<1,n.size()>>>(gn,firstLeaf,n.size(), gacc);
	
	// Copy accelerations back to CPU
	float *acc=new float[n.size()];
	check(hipMemcpy(acc,gacc,n.size()*sizeof(float),hipMemcpyDeviceToHost));
	
	// Print accelerations
	for (int i=firstLeaf;i<n.size();i++) {
		printf("Accel node %d, x=%.2f: %f\n", i, n[i].x, acc[i]);
	}
}


