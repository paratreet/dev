#include "hip/hip_runtime.h"
/**
  All-CUDA source code for 3D Ball-Hut
 */
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>

#include "ball1d_cudatree.h"

#define CUDA_USE_RECURSION // FIXME: use recursion

/**
  Kernel for GPU traversal
 */
__global__ void tryTraverse(const BallNodeData *treeNodes, int firstLeaf, int nTreeNodes, float *acc) {
  int tid = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
  int i = firstLeaf + tid;
  if (i < nTreeNodes) {
#ifdef CUDA_USE_RECURSION
    BallParaTree tree(treeNodes,firstLeaf,nTreeNodes);
#else
    BallParaTree untertree(treeNodes, firstLeaf, nTreeNodes);
    ManualStackTree<BallKey,typeof(untertree)> tree(untertree);
#endif
    BallKey treeRoot=1;
    BallConsumer<typeof(tree),BallKey> c(tree,treeNodes[i]);

    // Expand the tree root into the consumer
    tree.requestKey(treeRoot,c);

#ifndef CUDA_USE_RECURSION	
    tree.iterateToConsumer(c);
#endif

    acc[i] = c.acc;
  }
}

#define check(cudacall) { int err=cudacall; if (err!=hipSuccess) std::cout<<"CUDA ERROR "<<err<<" at line "<<__LINE__<<"'s "<<#cudacall<<"\n";}

/**
  Holder for nodes data
 */
BallNodeData *h_nodes;
BallKey firstLeaf;

inline bool isLeaf(int index) {
  return (index >= firstLeaf);
}

/**
  Recursively construct tree: not a member function 
  of BallParaTree, as deep copy would be necessary
  to pass a complete BallParaTree to the GPU.
  So BallParaTree is built instead on the GPU with
  the nodes data passed as an array from the CPU.
 */
void constructNodeArray(int index, float min, float max){
  // Interior node
  if (!isLeaf(index)) {
    float mid = (min+max)/2;
    h_nodes[index] = BallNodeData(20.0, mid, 0.0f, min, max);

    constructNodeArray(2*index, min, mid); // left child
    constructNodeArray(2*index+1, mid, max); // right child
  }
  // Leaf node
  else {
    float random = ((float) rand()) / (float) RAND_MAX;
    float pos = min + (max - min)*random;
    TRACE_BARNES(printf("[%d] Particle created : %6.2f\n",
          index, pos));

    h_nodes[index] = BallNodeData(20.0, pos, 25.0f, min, max);
  }
}

int main(int argc, char** argv) {
  // Parameters
  int depth = 3;
  if (argc >= 2) {
    depth = atoi(argv[1]);
  }

  // Record start time
  auto t1 = std::chrono::high_resolution_clock::now();

  // Calculate tree-related values
  int treeSize = (int)pow(2, depth+1);
  firstLeaf = (BallKey)pow(2, depth);
  int leafCount = (int)pow(2, depth-1);

  // Memory allocation on host
  h_nodes = (BallNodeData *)malloc(sizeof(BallNodeData) * treeSize);
  float *h_acc = (float*)malloc(sizeof(float) * treeSize);

  // Create nodes
  constructNodeArray(1, 0.0f, 100.0f);

  // Memory allocation on device
  float *d_acc;
  BallNodeData *d_nodes;
  check(hipMalloc((void **)&d_acc, sizeof(float) * treeSize));
  check(hipMalloc((void **)&d_nodes, sizeof(BallNodeData) * treeSize));

  // Copy nodes data to device
  check(hipMemcpy(d_nodes, h_nodes, sizeof(BallNodeData) * treeSize, hipMemcpyHostToDevice));

  // Accelerations for each leaf
  // Each leaf does top-down traversal on device
  if ((int)ceil(sqrt(leafCount+255/256)) >= 65536) {
    std::cout << "Too many nodes, grid size overflow" << std::endl;
    return -1;
  }
  dim3 blocks((int)ceil(sqrt(leafCount+255/256)), (int)ceil(sqrt(leafCount+255/256)), 0);
  tryTraverse<<<blocks,min(256,leafCount)>>>(d_nodes, firstLeaf, treeSize, d_acc);

  // Copy accelerations back to host
  check(hipMemcpy(h_acc, d_acc, sizeof(float) * treeSize, hipMemcpyDeviceToHost));

  // Print accelerations
  for (int i = firstLeaf; i < treeSize; i++) {
    TRACE_BARNES(printf("Accel node %d, pos=%6.2f: %f\n", i, h_nodes[i].pos, h_acc[i]));
  }

  // Free device memory
  check(hipFree(d_acc));
  check(hipFree(d_nodes));

  // Record end time
  auto t2 = std::chrono::high_resolution_clock::now();
  auto t_diff = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();

  // Free host memory
  free(h_nodes);
  free(h_acc);

  // Print time
  std::cout << "Execution time: " << t_diff << " ms" << std::endl;
}
