#include "hip/hip_runtime.h"
/**
  All-CUDA source code for 3D Barnes-Hut
 */
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include <hip/hip_runtime.h>

#include "barnes3d_cudatree.h"

#define CUDA_USE_RECURSION // FIXME: use recursion

/**
  Kernel for GPU traversal
 */
__global__ void tryTraverse(const BarnesNodeData *treeNodes, int firstLeaf, int nTreeNodes, float *acc) {
  int tid = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
  int i = firstLeaf + tid;
  if (i < nTreeNodes) {
#ifdef CUDA_USE_RECURSION
    BarnesParaTree tree(treeNodes,firstLeaf,nTreeNodes);
#else
    BarnesParaTree untertree(treeNodes, firstLeaf, nTreeNodes);
    ManualStackTree<BarnesKey,typeof(untertree)> tree(untertree);
#endif
    BarnesKey treeRoot=1;
    BarnesConsumer<typeof(tree),BarnesKey> c(tree,treeNodes[i]);

    // Expand the tree root into the consumer
    tree.requestKey(treeRoot,c);

#ifndef CUDA_USE_RECURSION	
    tree.iterateToConsumer(c);
#endif

    acc[i] = c.acc;
  }
}

#define check(cudacall) { int err=cudacall; if (err!=hipSuccess) std::cout<<"CUDA ERROR "<<err<<" at line "<<__LINE__<<"'s "<<#cudacall<<"\n";}

/**
  Holder for nodes data
 */
BarnesNodeData *h_nodes;
BarnesKey firstLeaf;

inline bool isLeaf(int index) {
  return (index >= firstLeaf);
}

/**
  Recursively construct tree: not a member function 
  of BarnesParaTree, as deep copy would be necessary
  to pass a complete BarnesParaTree to the GPU.
  So BarnesParaTree is built instead on the GPU with
  the nodes data passed as an array from the CPU.
 */
void constructNodeArray(int index, vector3d min, vector3d max){
  // Interior node
  if (!isLeaf(index)) {
    vector3d mid = (min+max)/2;
    h_nodes[index] = BarnesNodeData(20.0, mid, min, max);

    constructNodeArray(getChild(index, 0), vector3d(min.x,min.y,min.z), vector3d(mid.x,mid.y,mid.z));
    constructNodeArray(getChild(index, 1), vector3d(mid.x,min.y,min.z), vector3d(max.x,mid.y,mid.z));
    constructNodeArray(getChild(index, 2), vector3d(min.x,mid.y,min.z), vector3d(mid.x,max.y,mid.z));
    constructNodeArray(getChild(index, 3), vector3d(mid.x,mid.y,min.z), vector3d(max.x,max.y,mid.z));

    constructNodeArray(getChild(index, 4), vector3d(min.x,min.y,mid.z), vector3d(mid.x,mid.y,max.z));
    constructNodeArray(getChild(index, 5), vector3d(mid.x,min.y,mid.z), vector3d(max.x,mid.y,max.z));
    constructNodeArray(getChild(index, 6), vector3d(min.x,mid.y,mid.z), vector3d(mid.x,max.y,max.z));
    constructNodeArray(getChild(index, 7), vector3d(mid.x,mid.y,mid.z), vector3d(max.x,max.y,max.z));

  }
  // Leaf node
  else {
    float random = ((float) rand()) / (float) RAND_MAX;
    vector3d pos = min + (max - min)*random;
    TRACE_BARNES(printf("[%d] Particle created : (%6.2f, %6.2f, %6.2f)\n",
          index, pos.x, pos.y, pos.z));

    h_nodes[index] = BarnesNodeData(20.0, pos, min, max);
  }
}

int main(int argc, char** argv) {
  // Parameters
  int depth = 3;
  if (argc >= 2) {
    depth = atoi(argv[1]);
  }
  int treeSize = (int)pow(8, depth)/7 + 1;
  firstLeaf = (BarnesKey)((int)pow(8, depth)/56 + 1);
  int leafCount = (int)pow(8, depth-1);

  // Memory allocation on host
  h_nodes = (BarnesNodeData *)malloc(sizeof(BarnesNodeData) * treeSize);
  float *h_acc = (float*)malloc(sizeof(float) * treeSize);

  // Create nodes
  constructNodeArray(1, vector3d(0.0f, 0.0f, 0.0f), vector3d(100.0f, 100.0f, 100.0f));

  // Record start time
  auto t1 = std::chrono::high_resolution_clock::now();

  // Memory allocation on device
  float *d_acc;
  BarnesNodeData *d_nodes;
  check(hipMalloc((void **)&d_acc, sizeof(float) * treeSize));
  check(hipMalloc((void **)&d_nodes, sizeof(BarnesNodeData) * treeSize));

  // Copy nodes data to device
  check(hipMemcpy(d_nodes, h_nodes, sizeof(BarnesNodeData) * treeSize, hipMemcpyHostToDevice));

  // Accelerations for each leaf
  // Each leaf does top-down traversal on device
  if ((int)ceil(sqrt(leafCount+255/256)) >= 65536) {
    std::cout << "Too many nodes, grid size overflow" << std::endl;
    return -1;
  }
  dim3 blocks((int)ceil(sqrt(leafCount+255/256)), (int)ceil(sqrt(leafCount+255/256)), 0);
  tryTraverse<<<blocks,min(256,leafCount)>>>(d_nodes, firstLeaf, treeSize, d_acc);

  // Copy accelerations back to host
  check(hipMemcpy(h_acc, d_acc, sizeof(float) * treeSize, hipMemcpyDeviceToHost));

  // Print accelerations
  for (int i = firstLeaf; i < treeSize; i++) {
    TRACE_BARNES(printf("Accel node %d, pos=(%6.2f, %6.2f, %6.2f): %f\n", i, h_nodes[i].pos.x, h_nodes[i].pos.y, h_nodes[i].pos.z, h_acc[i]));
  }

  // Free device memory
  check(hipFree(d_acc));
  check(hipFree(d_nodes));

  // Record end time
  auto t2 = std::chrono::high_resolution_clock::now();
  auto t_diff = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();

  // Free host memory
  free(h_nodes);
  free(h_acc);

  // End timing
  std::cout << "Execution time: " << t_diff << " ms" << std::endl;
}
