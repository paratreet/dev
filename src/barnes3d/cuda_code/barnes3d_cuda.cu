#include "hip/hip_runtime.h"
/**
  All-CUDA source code for 3D Barnes-Hut
 */
#include <iostream>
#include <vector>
#include <cmath>
#include <hip/hip_runtime.h>

#include "barnes3d_cudatree.h"
//#define CUDA_USE_RECURSION

/**
  Kernel for GPU traversal
 */
__global__ void tryTraverse(const BarnesNodeData *treeNodes, int firstLeaf, int nTreeNodes, float *acc) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int i = firstLeaf + tid;

#ifdef CUDA_USE_RECURSION
  BarnesParaTree tree(treeNodes,firstLeaf,nTreeNodes);
#else
  BarnesParaTree untertree(treeNodes, firstLeaf, nTreeNodes);
  ManualStackTree<BarnesKey,typeof(untertree)> tree(untertree);
#endif
  BarnesKey treeRoot=1;
  BarnesConsumer<typeof(tree),BarnesKey> c(tree,treeNodes[i]);

  // Expand the tree root into the consumer
  tree.requestKey(treeRoot,c);

#ifndef CUDA_USE_RECURSION	
  tree.iterateToConsumer(c);
#endif

  acc[i] = c.acc;
}

#define check(cudacall) { int err=cudacall; if (err!=hipSuccess) std::cout<<"CUDA ERROR "<<err<<" at line "<<__LINE__<<"'s "<<#cudacall<<"\n";}

/**
  Holder for nodes data
 */
BarnesNodeData *h_nodes;
BarnesKey firstLeaf;
  
inline bool isLeaf(int index) {
  return (index >= firstLeaf);
}
	
/**
  Recursively construct tree: not a member function 
  of BarnesParaTree, as deep copy would be necessary
  to pass a complete BarnesParaTree to the GPU.
  So BarnesParaTree is built instead on the GPU with
  the nodes data passed as an array from the CPU.
 */
void constructNodeArray(int index, vector3d min, vector3d max){
  // Interior node
  if (!isLeaf(index)) {
    vector3d mid = (min+max)/2;
    h_nodes[index] = BarnesNodeData(20.0, mid, min, max);

    constructNodeArray(getChild(index, 0), vector3d(min.x,min.y,min.z), vector3d(mid.x,mid.y,mid.z));
    constructNodeArray(getChild(index, 1), vector3d(mid.x,min.y,min.z), vector3d(max.x,mid.y,mid.z));
    constructNodeArray(getChild(index, 2), vector3d(min.x,mid.y,min.z), vector3d(mid.x,max.y,mid.z));
    constructNodeArray(getChild(index, 3), vector3d(mid.x,mid.y,min.z), vector3d(max.x,max.y,mid.z));

    constructNodeArray(getChild(index, 4), vector3d(min.x,min.y,mid.z), vector3d(mid.x,mid.y,max.z));
    constructNodeArray(getChild(index, 5), vector3d(mid.x,min.y,mid.z), vector3d(max.x,mid.y,max.z));
    constructNodeArray(getChild(index, 6), vector3d(min.x,mid.y,mid.z), vector3d(mid.x,max.y,max.z));
    constructNodeArray(getChild(index, 7), vector3d(mid.x,mid.y,mid.z), vector3d(max.x,max.y,max.z));

  }
  // Leaf node
  else {
    float random = ((float) rand()) / (float) RAND_MAX;
    vector3d pos = min + (max - min)*random;
    printf("[%d] Particle created : (%6.2f, %6.2f, %6.2f)\n",
          index, pos.x, pos.y, pos.z);

      h_nodes[index] = BarnesNodeData(20.0, pos, min, max);
  }
}

int main(int argc, char** argv) {
  int depth = 3;
  if (argc == 2) {
    depth = atoi(argv[1]);
  }
  int treeSize = (int)pow(8, depth)/7 + 1;
  firstLeaf = (BarnesKey)((int)pow(8, depth)/56 + 1);
  int leafCount = (int)pow(8, depth-1);

  // Create nodes on CPU
  h_nodes = (BarnesNodeData *)malloc(sizeof(BarnesNodeData) * treeSize);
  constructNodeArray(1, vector3d(0.0f, 0.0f, 0.0f), vector3d(100.0f, 100.0f, 100.0f));

  // Copy nodes data to GPU
  BarnesNodeData *d_nodes;
  check(hipMalloc((void **)&d_nodes, sizeof(BarnesNodeData) * treeSize));
  check(hipMemcpy(d_nodes, h_nodes, sizeof(BarnesNodeData) * treeSize, hipMemcpyHostToDevice));

  // Accelerations for each leaf
  float *d_acc;
  check(hipMalloc((void **)&d_acc, sizeof(float) * treeSize));

  // Each leaf does top-down traversal on GPU
  tryTraverse<<<(leafCount+255)/256,min(256,leafCount)>>>(d_nodes, firstLeaf, treeSize, d_acc);

  // Copy accelerations back to CPU
  float *h_acc = new float[treeSize];
  check(hipMemcpy(h_acc, d_acc, sizeof(float) * treeSize, hipMemcpyDeviceToHost));

  hipDeviceSynchronize();
  printf("synchronized\n");

  // Print accelerations
  for (int i = firstLeaf; i < treeSize; i++) {
    printf("Accel node %d, pos=(%6.2f, %6.2f, %6.2f): %f\n", i, h_nodes[i].pos.x, h_nodes[i].pos.y, h_nodes[i].pos.z, h_acc[i]);
  }
}


